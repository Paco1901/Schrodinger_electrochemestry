#include "hip/hip_runtime.h"
function X = SimulateDeposition(sys_length, t_max, deposition_rate, rule)
format long;
L = sys_length;
rate = deposition_rate;
temp_max = 0; % dummy variable
MaxHeight = 100;
for i = 1:L
    h(i) = 0;
end
for i = 1:t_max
    width(i) = 0;
end
% generating an array of random numbers
A =  ceil(L*rand(t_max,rate));
% the simulation
if((rule == 1) | (rule == 2))
    if (rule == 1)
        rule_name = 'Simple';
    else
        rule_name = 'Ballistic';
    end
for t = 1: t_max
    for k = 1 : rate
            i = A(t,k);
        if (rule == 1)
            h(i) = h(i) + 1;
            hit(i,h(i)) = 1;
        else
            if (i == 1)
                h(i) = max(h(i)+1,h(i+1));
                hit(i,h(i)) = 1;
            elseif (i == L)
                h(i) = max(h(i-1),h(i)+1);
                hit(i,h(i)) = 1;
            else    
                temp_max = max(h(i-1),h(i+1));
                h(i) = max(temp_max,h(i)+1);
                hit(i,h(i)) = 1;
            end
        end    
    end
    
    width(t) = std(h,1); 
    
    % Printing h(i,t) on the screen to verify the results
    % (note: You can uncomment the following 3 lines if you want to check)
    % fprintf('Time : %d\n', t);
    % fprintf('Width Fluctuation : %d\n', width(t));
    % h
end
% Plotting the width as a function of time using semilogx function
figure('Name','Width Fluctuation as a Function of Time in 1 Dimensional Lattice');
semilogx(width(1:t_max),'s',...
                               'MarkerEdgeColor','r',...
                               'MarkerFaceColor','b',...
                               'MarkerSize',4)
xlabel('Log10(Time)','FontSize',12)
ylabel('W(L,t)','FontSize',12)
title(['L = ',num2str(L),'   Time Max = ',num2str(t_max), '   Rate = ', num2str(rate), '   Rule = ', rule_name],'FontSize',12)
grid on
print -djpeg WidthFluc_1D.jpg;
% Plotting the surface
figure
for iatom=1:L
   for jatom=h(iatom):-1:1
       if (hit(iatom,jatom)==1)
        plotatom(iatom,jatom,'b')
       end
       hold on
   end
end
axis([1 L 0 (max(h)+10)])
xlabel('column i','FontSize',12)
ylabel('h(i)','FontSize',12)
title(['Surface Plot of L = ', num2str(L)],'FontSize',12)
print -djpeg Surface_Plot1D.jpg;
else
fprintf('\nInvalid Rule\n');
fprintf('\nRule = 1 for Simple Rule where h(i,t+1) = h(i,t) + 1\n');
fprintf('\nRule = 2 for Ballistic Rule where h(i,t+1) = MAX[h(i-1,t),h(i,t)+1,h(i+1,t)]\n');
end
%-------------------------------------------------------------------------%
function plotatom(x0,y0,color)
% plot one atom in the specified color, centered at (x0,y0) with radius r
r = 0.5;
theta = 0:0.1:6.29;
x = x0 + r*cos(theta);
y = y0 + r*sin(theta);
fill(x,y,color)
plot(x,y,color)
