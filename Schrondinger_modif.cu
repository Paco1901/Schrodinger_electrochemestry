#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include <cstdlib>
#include <complex>
 
using namespace std;
typedef complex<long double> long_double_complex;
long_double_complex Il=long_double_complex(0.0,1.0);
 
const long double Cl_SI_HBAR           =1.054571596e-34;
const long double Cl_SI_H=1.054571596e-34*2.0*M_PI;
const long double Cl_SI_MASS_ELECTRON  =9.10938188e-31;
const long double Cl_SI_CHARGE_ELECTRON=1.602176462e-19;
const long double Cl_SI_EPSILON0       =8.854187817e-12;
const long double Cl_SI_KB             = 1.3806503e-23;
 
#ifdef _GPU
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#else
#include <fftw3.h>
#endif
 
#ifdef _GPU
 
#define BLOCK_DIM  16
#define BLOCK_SIZE 256
 
__global__ void GPUMatrixElementMult(double2* iA, double2* iB, double2* oC, int N, double scale)
{
   const int idx = blockIdx.x * blockDim.x + threadIdx.x;
   double2 z;
   if (idx<N)
   {
      z.x = iA[idx].x * iB[idx].x - iA[idx].y * iB[idx].y;
      z.y = iA[idx].x * iB[idx].y + iA[idx].y * iB[idx].x;
      oC[idx].x = z.x *scale;
      oC[idx].y = z.y *scale;
   }
}
 
/* GPU Correlation requires to adapt the "reduction" example from the SDK,
since we have to avoid memory synchronization problems when we write the results
from all threads in a single double */
 
/* slow serial kernel */
__global__ void  GPUSummation_serial( const double2  *iA, double2* oC, int N, int it )
{
    if( threadIdx.x == 0 )
    {
        oC[it].x = 0.0;
        oC[it].y = 0.0;
        for(int idx = 0; idx < N; idx++)
        {
            oC[it].x += iA[idx].x;
            oC[it].y += iA[idx].y;
        }
    }
}
 
__global__ void  GPUSummation_parallel_partial( const double2  *iA, double2* g_odata, int N )
{
    __shared__ double2 sdata[BLOCK_SIZE];
 
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    unsigned int gridSize = BLOCK_SIZE * gridDim.x;
    double2 accum;
    
    accum.x = iA[idx].x;
    accum.y = iA[idx].y;
    idx+=gridSize;
 
    while (idx < N)
    {
        accum.x += iA[idx].x;
        accum.y += iA[idx].y;
        idx += gridSize;
    }
 
    sdata[tid].x=accum.x;
    sdata[tid].y=accum.y;
 
    __syncthreads();
 
    if (BLOCK_SIZE >= 512) { if (tid < 256) 
    { sdata[tid].x += sdata[tid + 256].x; sdata[tid].y += sdata[tid + 256].y; } __syncthreads(); }
    if (BLOCK_SIZE >= 256) { if (tid < 128) 
    { sdata[tid].x += sdata[tid + 128].x; sdata[tid].y += sdata[tid + 128].y; } __syncthreads(); }
    if (BLOCK_SIZE >= 128) { if (tid <  64) 
    { sdata[tid].x += sdata[tid +  64].x; sdata[tid].y += sdata[tid +  64].y; } __syncthreads(); }
 
    if (tid < 32)
    {
        if (BLOCK_SIZE >=  64) { sdata[tid].x += sdata[tid + 32].x; sdata[tid].y += sdata[tid + 32].y;  __syncthreads(); }
        if (BLOCK_SIZE >=  32) { sdata[tid].x += sdata[tid + 16].x; sdata[tid].y += sdata[tid + 16].y;  __syncthreads(); }
        if (BLOCK_SIZE >=  16) { sdata[tid].x += sdata[tid +  8].x; sdata[tid].y += sdata[tid +  8].y;  __syncthreads(); }
        if (BLOCK_SIZE >=   8) { sdata[tid].x += sdata[tid +  4].x; sdata[tid].y += sdata[tid +  4].y;  __syncthreads(); }
        if (BLOCK_SIZE >=   4) { sdata[tid].x += sdata[tid +  2].x; sdata[tid].y += sdata[tid +  2].y;  __syncthreads(); }
        if (BLOCK_SIZE >=   2) { sdata[tid].x += sdata[tid +  1].x; sdata[tid].y += sdata[tid +  1].y;  __syncthreads(); }
    }
    // write result for this block to global mem 
    if (tid == 0) 
    {
       g_odata[blockIdx.x].x = sdata[0].x; 
       g_odata[blockIdx.x].y = sdata[0].y; 
    }
}
 
#else
 
// Store in oC the matrix element wise multiplication of iA and iC, all objects have N elements, scale the result
void CPUMatrixElementMultScale(fftw_complex* iA, fftw_complex* iB, fftw_complex* oC, int N, double scale)
{
   int idx;
   fftw_complex z;
   for(idx=0;idx<N;idx++)
   {
      z[0] = iA[idx][0] * iB[idx][0] - iA[idx][1] * iB[idx][1];
      z[1] = iA[idx][0] * iB[idx][1] + iA[idx][1] * iB[idx][0];
      oC[idx][0] = z[0] * scale;
      oC[idx][1] = z[1] * scale;
   }
}
 
void CPUSummation(fftw_complex* iA, fftw_complex* oC, int N, int it)
{
   int idx;
 
   oC[it][0]=0.0;
   oC[it][1]=0.0;
 
   for(idx=0;idx<N;idx++)
   {
      oC[it][0] +=  iA[idx][0];
      oC[it][1] +=  iA[idx][1];
   }
}
 
#endif
 
long double psiInit(long double x, long double y, long double a )
{
   return exp(-(x*x+y*y)/(2.0*a*a))/(a*sqrt(M_PI));
}
 
long double potential( long double x, long double y )
{
   return 10000.0*Cl_SI_CHARGE_ELECTRON*x;
}
 
int main( void )
{
   int NT,NX,NY,N,in,it;
   double *xGrid,*yGrid,*kxGrid,*kyGrid,*kxGridShift,*kyGridShift;
   long double x0,y0,x1,y1,DX,DY,dx,dy;
   long double dt,h,m,a,meff,err;
   FILE *fd;
 
#ifdef _GPU
   hipfftHandle plan;
#else
   fftw_plan  fftplanAll;
   fftw_plan ifftplanAll;
#endif
 
#ifdef _GPU
   double2 *dev_psiPosGrid, *dev_psiPosInitGrid, *dev_psiMomGrid, *dev_expTGrid, *dev_expVGrid, *dev_act ,*dev_psiCor,*dev_g_odata; // device
   double2 *psiPosGrid,*psiPosInitGrid,*psiMomGrid,*expTGrid,*expVGrid,*act,*psiCor;
#else
   fftw_complex *psiPosGrid,*psiPosInitGrid,*psiMomGrid,*expTGrid,*expVGrid,*act,*psiCor;
#endif
 
   size_t sizeN,sizeNT;
   clock_t c0,c1;
 
   NX=1024; // grid points in x-direction
   NY=1024; // grid points in y-direction
 
   DX=0.4E-6; // half-width of potential in x-direction
   DY=0.4E-6; // half-width of potential in y-direction
 
   dx=2.0*DX/(double)(NX); // grid step size in x-direction
   dy=2.0*DY/(double)(NY); // grid step size in y-direction
   x0=-DX; // lower left corner x-coordinate
   y0=-DY; // lower left corner y-coordinate
   x1=x0+2.0*DX; // upper right corner x-coordinate
   y1=y0+2.0*DY; // upper right corner y-coordinate
   N=NX*NY; // total number of grid points
 
   NT=1000; // number of time-propagtion steps
   dt=100.0E-15; // time step
 
   meff=0.067; // effective mass
   a=80.0E-9; // gaussian width of initial wavepacket
   h=Cl_SI_HBAR; // hbar
   m=meff*Cl_SI_MASS_ELECTRON; // electron mass in kg
 
#ifdef _GPU
   sizeN  = N  * sizeof(double2);
   sizeNT = NT * sizeof(double2);
   psiPosGrid     = (double2*)malloc(sizeN);
   psiPosInitGrid = (double2*)malloc(sizeN);
   psiMomGrid     = (double2*)malloc(sizeN);
   psiCor         = (double2*)malloc(sizeN);
   expTGrid       = (double2*)malloc(sizeN);
   expVGrid       = (double2*)malloc(sizeN);
   act            = (double2*)malloc(sizeNT);
 
   hipMalloc((void**)&dev_psiPosGrid,sizeN);
   hipMalloc((void**)&dev_psiPosInitGrid,sizeN);
   hipMalloc((void**)&dev_psiMomGrid,sizeN);
   hipMalloc((void**)&dev_psiCor,sizeN);
   hipMalloc((void**)&dev_expTGrid,sizeN);
   hipMalloc((void**)&dev_expVGrid,sizeN);
   hipMalloc((void**)&dev_act,sizeNT);
   hipMalloc((void**)&dev_g_odata,BLOCK_SIZE);
   hipfftPlan2d(&plan, NX, NY, HIPFFT_Z2Z);
#else
   sizeN  = N  * sizeof(fftw_complex);
   sizeNT = NT * sizeof(fftw_complex);
   psiPosGrid     = (fftw_complex*)fftw_malloc(sizeN);
   psiPosInitGrid = (fftw_complex*)fftw_malloc(sizeN);
   psiMomGrid     = (fftw_complex*)fftw_malloc(sizeN);
   psiCor         = (fftw_complex*)fftw_malloc(sizeN);
   expTGrid       = (fftw_complex*)fftw_malloc(sizeN);
   expVGrid       = (fftw_complex*)fftw_malloc(sizeN);
   act            = (fftw_complex*)fftw_malloc(sizeNT);
 
   fftplanAll =fftw_plan_dft_2d(NX,NY,(fftw_complex*)psiPosGrid,(fftw_complex*)psiMomGrid,FFTW_BACKWARD,FFTW_MEASURE);
   ifftplanAll=fftw_plan_dft_2d(NX,NY,(fftw_complex*)psiMomGrid,(fftw_complex*)psiPosGrid,FFTW_FORWARD ,FFTW_MEASURE);
#endif
 
   // initialize the position space grid
   // initialize the momentum space grid and shift it
   xGrid      = (double*) malloc(sizeof(double)*NX);
   kxGrid     = (double*) malloc(sizeof(double)*NX);
   kxGridShift= (double*) malloc(sizeof(double)*NX);
   kyGrid     = (double*) malloc(sizeof(double)*NY);
   kyGridShift= (double*) malloc(sizeof(double)*NY);
   yGrid      = (double*) malloc(sizeof(double)*NY);
   for(int ix=0;ix<NX;ix++)
   {
      xGrid[ix]=x0+ix*dx;
      kxGrid[ix]=-M_PI/dx+double(ix)*2.0*M_PI/double(NX)/dx;
   }
   for(int ix=0;ix<NX/2;ix++)
   {
      kxGridShift[ix]=kxGrid[NX/2+ix];
   }
   for(int ix=NX/2;ix<NX;ix++)
   {
      kxGridShift[ix]=kxGrid[ix-NX/2];
   }
   for(int iy=0;iy<NY;iy++)
   {
      yGrid[iy]=y0+iy*dy;
      kyGrid[iy]=-M_PI/dy+double(iy)*2.0*M_PI/double(NY)/dy;
   }
   for(int iy=0;iy<NY/2;iy++)
   {
      kyGridShift[iy]=kyGrid[NY/2+iy];
   }
   for(int iy=NY/2;iy<NY;iy++)
   {
      kyGridShift[iy]=kyGrid[iy-NY/2];
   }
 
   for(int iy=0;iy<NY;iy++)
   {
      for(int ix=0;ix<NX;ix++)
      {
         int in=ix*NY+iy;
         // do all intermediate calculations in long double to avoid any out of range errors, which DO happen if one uses double for the exp()
         long double V=potential(xGrid[ix],yGrid[iy]);
         long_double_complex expV=exp(Il*(long double)(-(V                                                                                        )*dt/h));
         long_double_complex expT=exp(Il*(long double)(-(h*h*kxGridShift[ix]*kxGridShift[ix]/(2.0l*m)+h*h*kyGridShift[iy]*kyGridShift[iy]/(2.0l*m))*dt/h));
         long_double_complex psi=psiInit(xGrid[ix],yGrid[iy],a);
         // demote long double results to double
#ifdef _GPU
         expVGrid[in].x=expV.real();
         expVGrid[in].y=expV.imag();
         expTGrid[in].x=expT.real();
         expTGrid[in].y=expT.imag();
         psiPosGrid[in].x=(double)psi.real();
         psiPosGrid[in].y=(double)psi.imag();
         psiPosInitGrid[in].x=(double)psi.real();
         psiPosInitGrid[in].y=(double)psi.imag();
#else
         expVGrid[in][0]=(double)expV.real();
         expVGrid[in][1]=(double)expV.imag();
         expTGrid[in][0]=(double)expT.real();
         expTGrid[in][1]=(double)expT.imag();
         psiPosGrid[in][0]=(double)psi.real();
         psiPosGrid[in][1]=(double)psi.imag();
         psiPosInitGrid[in][0]=(double)psi.real();
         psiPosInitGrid[in][1]=(double)psi.imag();
#endif
      }
   }
 
#ifdef _GPU
   for(int it=0;it<NT;it++)
   {
      act[it].x=0.0;
      act[it].y=0.0;
   }
   hipMemcpy(dev_psiPosGrid,psiPosGrid,sizeN,hipMemcpyHostToDevice);
   hipMemcpy(dev_psiMomGrid,psiMomGrid,sizeN,hipMemcpyHostToDevice);
   hipMemcpy(dev_psiPosInitGrid,psiPosInitGrid,sizeN,hipMemcpyHostToDevice);
   hipMemcpy(dev_expTGrid,expTGrid,sizeN,hipMemcpyHostToDevice);
   hipMemcpy(dev_expVGrid,expVGrid,sizeN,hipMemcpyHostToDevice);
   hipMemcpy(dev_act,act,sizeNT,hipMemcpyHostToDevice);
#endif
 
   fd=fopen("result_ini.dat","w");
   for(in=0;in<N;in+=100)
   {
#ifdef _GPU
      fprintf(fd,"GPU psiPosInitGrid[%i]=(%e,%e)\n",in,(double)psiPosInitGrid[in].x,(double)psiPosInitGrid[in].y);
#else
      fprintf(fd,"CPU psiPosInitGrid[%i]=(%e,%e)\n",in,(double)psiPosInitGrid[in][0],(double)psiPosInitGrid[in][1]);
#endif
   }
   fclose(fd);
 
   fprintf(stderr,"Initializing finished. Starting timer ...\n");
 
   c0=clock();
   for(it=0;it<NT;it++)
   {
#ifdef _GPU
      GPUMatrixElementMult<<<N/256,256>>>(dev_expVGrid,dev_psiPosGrid,dev_psiPosGrid,N,1.0);
      hipDeviceSynchronize();
      hipfftExecZ2Z(plan, dev_psiPosGrid, dev_psiMomGrid, HIPFFT_BACKWARD);
      hipDeviceSynchronize();
      GPUMatrixElementMult<<<N/256,256>>>(dev_expTGrid,dev_psiMomGrid,dev_psiMomGrid,N,1.0/(double)N);
      hipDeviceSynchronize();
      hipfftExecZ2Z(plan, dev_psiMomGrid, dev_psiPosGrid, HIPFFT_FORWARD);
      hipDeviceSynchronize();
      GPUMatrixElementMult<<<N/256,256>>>(dev_psiPosGrid,dev_psiPosInitGrid,dev_psiCor,N,1.0);
      hipDeviceSynchronize();
      GPUSummation_parallel_partial<<<BLOCK_SIZE,BLOCK_SIZE>>>(dev_psiCor,dev_g_odata,(unsigned int)N);
      hipDeviceSynchronize();
      GPUSummation_serial<<<1,1>>>(dev_g_odata,dev_act,BLOCK_SIZE,it);
      hipDeviceSynchronize();
#else
      CPUMatrixElementMultScale(expVGrid,psiPosGrid,psiPosGrid,N,1.0);
      fftw_execute(fftplanAll);
      CPUMatrixElementMultScale(expTGrid,psiMomGrid,psiMomGrid,N,1.0/(double)N);
      fftw_execute(ifftplanAll);
      CPUMatrixElementMultScale(psiPosGrid,psiPosInitGrid,psiCor,N,1.0);
      CPUSummation(psiCor,act,N,it);
#endif
   }
   c1=clock();
 
   fprintf(stderr,"Propagation took %.2f s\n",(double)(c1-c0)/(double)CLOCKS_PER_SEC);
 
#ifdef _GPU
   hipMemcpy(act,dev_act,sizeNT,hipMemcpyDeviceToHost);
   hipMemcpy(psiPosGrid,dev_psiPosGrid,sizeN,hipMemcpyDeviceToHost);
 
   fd=fopen("result_gpu_act_dp.dat","w");
   // write recorded autocorrelation function at each time-step
   for(it=0;it<NT;it++)
   {
      fprintf(fd,"%e %e %e\n",(double)(it*dt),(double)(act[it].x*dx*dy),(double)(act[it].y*dx*dy));
   }
   fclose(fd);
#else
   fd=fopen("result_cpu_act_dp.dat","w");
   // write recorded autocorrelation function at each time-step
   for(it=0;it<NT;it++)
   {
      fprintf(fd,"%e %e %e\n",(double)(it*dt),(double)(act[it][0]*dx*dy),(double)(act[it][1]*dx*dy));
   }
   fclose(fd);
#endif
 
   // all memory frees are missing ...
 
   return 0;
}
